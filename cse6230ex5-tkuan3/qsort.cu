#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/random.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/tuple.h>
#include <thrust/execution_policy.h>
#include <hiprand.h>

#define DEBUG 0

typedef thrust::tuple<int, int> iter_pair;
typedef thrust::tuple<int &, int &> iter_pair_ref;
typedef thrust::tuple<int &, int &, int &> iter_trip_ref;
typedef thrust::tuple<int &, int &, int &, int &> iter_quart_ref;
typedef thrust::tuple<int &, int &, int &, int &, int &> iter_quint_ref;

/// fill the array with random numbers
void generate(thrust::device_vector<int> &array) {
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 0);
  int * A_start_raw = (&array[0]).get();
  hiprandGenerate(gen, (unsigned int *) A_start_raw, array.size());
} 

/// fill the array with 0..N
void generate_sorted(thrust::device_vector<int> &array) {
  thrust::counting_iterator<int> first(0);
  thrust::copy(first, first + array.size(), array.begin());
}

/// test if sorted
bool is_sorted(thrust::device_vector<int> &array) {
  size_t N = array.size();
  if (N < 2) {
    return true;
  }
  // get references to the first and second elements
  auto first = array.begin();
  auto second = next(first);
  // get references to the second-to-last and last elements
  auto last = array.end();
  auto beforelast = prev(last);

  // loop over consecutive pairs by making a zip iterator
  // that looks at pairs (array[0], array[1]), (array[1], array[2]),
  //
  // each pair (array[i], array[i+1]) evaluates to true
  // if array[i] <= array[i+1]
  // 
  // the list is sorted if this is true for all pairs, so we
  // reduce the {true,false} values with a logical and
  bool sorted = thrust::transform_reduce(
      thrust::device,
      thrust::make_zip_iterator(thrust::make_tuple(first, second)),
      thrust::make_zip_iterator(thrust::make_tuple(beforelast, last)),
      [] __device__ (iter_pair pair) -> bool {
        return thrust::get<0>(pair) <= thrust::get<1>(pair);
      },
      true,
      thrust::logical_and<bool>());
  return sorted;
}

// comparator swap
template<typename T>
void __host__ __device__ cswp(T &a1, T &a2) {
  T min = a1 <= a2 ? a1 : a2;
  T max = a1 <= a2 ? a2 : a1;
  a1 = min;
  a2 = max;
}

// see <https://stackoverflow.com/questions/3903086/standard-sorting-networks-for-small-values-of-n>
template<typename T>
void __host__ __device__ sort_3(T &a1, T &a2, T &a3) {
  cswp<T>(a1, a2);
  cswp<T>(a1, a3);
  cswp<T>(a2, a3);
}

template<typename T>
void __host__ __device__ sort_4(T &a1, T &a2, T &a3, T &a4) {
  cswp<T>(a1, a2); cswp<T>(a3, a4);
  cswp<T>(a1, a3); cswp<T>(a2, a4);
  cswp<T>(a2, a3);
}

template<typename T>
void __host__ __device__ sort_5(T &a1, T &a2, T &a3, T &a4, T &a5) {
  cswp<T>(a1, a2); cswp<T>(a3, a4);
  cswp<T>(a1, a3); cswp<T>(a2, a5);
  cswp<T>(a1, a2); cswp<T>(a3, a4);
  cswp<T>(a2, a3); cswp<T>(a4, a5);
  cswp<T>(a3, a4);
}

/// partition so that the kth element (by order) is in position k
void select_host(int *A, int *A_end, size_t k) {
  size_t n = A_end - A;
  size_t n_div_5 = n / 5;
  size_t n_rem_5 = n % 5;

  int *subA[5];
  size_t offsets[6];

  offsets[0] = 0;

  // break up A into 5 sub-array of equal length
  for (size_t i = 0; i < 5; i++) {
    subA[i] = &A[i * n_div_5 + (i * n_rem_5) / 5];
    offsets[i+1] = (i+1) * n_div_5 + ((i+1) * n_rem_5) / 5;
  }

#if DEBUG
  std::cout << "Pre " << std::endl;
  for (size_t j = 0; j < 5; j++) {
    for (size_t i = offsets[j]; i < offsets[j+1]; i++) {
      std::cout << A[i] << " ";
    }
    std::cout << std::endl;
  }
#endif

  // sort matching elements of the five arrays
  for (size_t j = 0; j < n_div_5; j++) {
    sort_5<int>(subA[0][j], subA[1][j], subA[2][j], subA[3][j], subA[4][j]);
  }

  // handle the remainders when some of the arrays are longer than others
  switch (n_rem_5) {
  case 4:
    sort_4<int>(subA[1][n_div_5], subA[2][n_div_5], subA[3][n_div_5], subA[4][n_div_5]);
    break;
  case 3:
    sort_3<int>(subA[1][n_div_5], subA[3][n_div_5], subA[4][n_div_5]);
    break;
  case 2:
    cswp<int>(subA[2][n_div_5], subA[4][n_div_5]);
    break;
  default:
    break;
  }

#if DEBUG
  std::cout << "After remainder " << std::endl;
  for (size_t j = 0; j < 5; j++) {
    for (size_t i = offsets[j]; i < offsets[j+1]; i++) {
      std::cout << A[i] << " ";
    }
    std::cout << std::endl;
  }
#endif

  if (n <= 5) {
    return;
  }

  // which array should have element k?
  size_t subindex = 0;
  if (k >= offsets[1]) {subindex++;}
  if (k >= offsets[2]) {subindex++;}
  if (k >= offsets[3]) {subindex++;}
  if (k >= offsets[4]) {subindex++;}

  // what should its location be?
  size_t subn = offsets[subindex+1] - offsets[subindex];
  size_t subk = k - offsets[subindex];
  select_host(subA[subindex], subA[subindex] + subn, subk);
  int pivot = subA[subindex][subk];

  // divide the partition into less than pivot and greater or equal to the pivot
  int *A_l = A;
  int *A_eq = thrust::partition(
      thrust::host,
      A, A_end,
      [pivot] __host__ (int v) -> bool {
        return v < pivot;
      }
      );
  int *A_g = thrust::partition(
      thrust::host,
      A_eq, A_end,
      [pivot] __host__ (int v) -> bool {
        return v == pivot;
      }
      );

  size_t n_lo = A_eq - A_l;
  size_t n_eq = A_g - A_eq;
  size_t n_hi = A_end - A_g;

  if (n_lo <= k && k < n_lo + n_eq) {
    return;
  } else if (k < n_lo) {
    select_host(A_l, A_l + n_lo, k);
  } else {
    select_host(A_g, A_g + n_hi, k - (n_lo + n_eq));
  }
#if DEBUG
  std::cout << "Final " << std::endl;
  for (size_t i = 0; i < k; i++) {
    std::cout << A[i] << " ";
  }
  std::cout << std::endl;
  std::cout << A[k] << std::endl;
  for (size_t i = k+1; i < n; i++) {
    std::cout << A[i] << " ";
  }
  std::cout << std::endl;
#endif
}

void select_device(thrust::device_ptr<int> A, thrust::device_ptr<int> A_end, size_t k) {
  size_t n = A_end - A;
  size_t n_div_5 = n / 5;
  size_t n_rem_5 = n % 5;
  
  thrust::device_ptr<int> subA[5];
  size_t offsets[6];

  offsets[0] = 0;

  // break up A into 5 sub-array of equal length
  for (size_t i = 0; i < 5; i++) {
    subA[i] = &A[i * n_div_5 + (i * n_rem_5) / 5];
    offsets[i+1] = (i+1) * n_div_5 + ((i+1) * n_rem_5) / 5;
  }

  // sort matching elements of the five arrays
  thrust::for_each_n(
    thrust::device,
    thrust::make_zip_iterator(thrust::make_tuple(&subA[0][0], &subA[1][0], &subA[2][0], &subA[3][0], &subA[4][0])),
    n_div_5,
    [] __device__ (auto pair) -> void {
      sort_5<int>(thrust::get<0>(pair), thrust::get<1>(pair), thrust::get<2>(pair), thrust::get<3>(pair), thrust::get<4>(pair));
    }
  );

  // handle the remainders when some of the arrays are longer than others
  thrust::for_each_n(
    thrust::device,
    thrust::make_zip_iterator(thrust::make_tuple(&subA[0][n_div_5], &subA[1][n_div_5], &subA[2][n_div_5], &subA[3][n_div_5], &subA[4][n_div_5])),
    1,
    [n_rem_5] __device__ (auto pair) -> void {
      switch (n_rem_5) {
        case 4:
          sort_4<int>(thrust::get<1>(pair), thrust::get<2>(pair), thrust::get<3>(pair), thrust::get<4>(pair));
          break;
        case 3:
          sort_3<int>(thrust::get<1>(pair), thrust::get<3>(pair), thrust::get<4>(pair));
          break;
        case 2:
          cswp<int>(thrust::get<2>(pair), thrust::get<4>(pair));
          break;
        default:
          break;
      }
    }
  );

  if (n <= 5) {
    return;
  }

  // which array should have element k?
  size_t subindex = 0;
  if (k >= offsets[1]) {subindex++;}
  if (k >= offsets[2]) {subindex++;}
  if (k >= offsets[3]) {subindex++;}
  if (k >= offsets[4]) {subindex++;}

  // what should its location be?
  size_t subn = offsets[subindex+1] - offsets[subindex];
  size_t subk = k - offsets[subindex];
  select_device(subA[subindex], subA[subindex] + subn, subk);
  //int pivot = subA[subindex][subk];
  thrust::device_ptr<int> pivot = thrust::device_malloc<int>(1);
  *pivot = subA[subindex][subk];

  // divide the partition into less than pivot and greater or equal to the pivot
  thrust::device_ptr<int> A_l = A;
  thrust::device_ptr<int> A_eq = thrust::partition(
      thrust::device,
      A, A_end,
      [pivot] __device__ (int v) -> bool {
        return v < *pivot;
      }
      );
  thrust::device_ptr<int> A_g = thrust::partition(
      thrust::device,
      A_eq, A_end,
      [pivot] __device__ (int v) -> bool {
        return v == *pivot;
      }
      );
  
  thrust::device_free(pivot);

  size_t n_lo = A_eq - A_l;
  size_t n_eq = A_g - A_eq;
  size_t n_hi = A_end - A_g;

  if (n_lo <= k && k < n_lo + n_eq) {
    return;
  } else if (k < n_lo) {
    select_device(A_l, A_l + n_lo, k);
  } else {
    select_device(A_g, A_g + n_hi, k - (n_lo + n_eq));
  }
}

void select(thrust::device_ptr<int> A, thrust::device_ptr<int> A_end, size_t k) {
#if 0
  size_t n = A_end - A;

  thrust::host_vector<int> A_host(n);

  thrust::copy(A, A_end, &A_host[0]);

  select_host(&A_host[0], &A_host[0] + n, k);

  thrust::copy(&A_host[0], &A_host[0] + n, A);
#else
  select_device(A, A_end, k);
#endif
}

void qsort(thrust::device_ptr<int> A, thrust::device_ptr<int> A_end) {
  size_t n = A_end - A;

  if (n <= 2) {
    return;
  }
  select(A, A_end, n / 2);
  qsort(A, &A[n/2]);
  qsort(&A[n/2], A_end);
}

int main(int argc, char **argv) {
  size_t N = 1000000;

  if (argc > 1) {
    N = (size_t) strtoull(argv[1], NULL, 10);
  }
  std::cout << "Sorting " << N << " ints" << std::endl;

  thrust::device_vector<int> array(N);
  generate(array);

  bool input_sorted = is_sorted(array);
  std::cout << "Input is " << (input_sorted ? "" : "not ") << "sorted" << std::endl;

  qsort(&array[0], &array[N]);

  bool output_sorted = is_sorted(array);
  std::cout << "Output is " << (output_sorted ? "" : "not ") << "sorted" << std::endl;
  return 0;
}
